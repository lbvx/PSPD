
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to add two vectors
__global__ void vectorAddition(float *a, float *b, float *result, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    // Size of the vectors
    int size = 1000000;

    // Allocate memory for vectors on the host
    float *h_a, *h_b, *h_result;
    h_a = (float*)malloc(size * sizeof(float));
    h_b = (float*)malloc(size * sizeof(float));
    h_result = (float*)malloc(size * sizeof(float));

    // Initialize vectors on the host
    for (int i = 0; i < size; ++i) {
        h_a[i] = (float) i;
        h_b[i] = (float) i * 2.0f;
    }

    // Allocate memory for vectors on the device
    float *d_a, *d_b, *d_result;
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_b, size * sizeof(float));
    hipMalloc((void**)&d_result, size * sizeof(float));

    // Copy vectors from host to device
    hipMemcpy(d_a, h_a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(float), hipMemcpyHostToDevice);

    // Set up grid and block sizes
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    // Launch the vector addition kernel
    vectorAddition<<<numBlocks, blockSize>>>(d_a, d_b, d_result, size);

    // Copy result vector from device to host
    hipMemcpy(h_result, d_result, size * sizeof(float), hipMemcpyDeviceToHost);

    // Verify the results
    for (int i = 0; i < size; ++i) {
        printf("%f ", h_result[i]);
    }
    printf("\n");

    printf("Vector addition on GPU completed successfully!\n");

    // Free memory
    free(h_a);
    free(h_b);
    free(h_result);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
