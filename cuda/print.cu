
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA kernel to add two numbers
__global__ void addKernel(float a, float b, float *result) {
    *result = a + b;
    printf("blockId = %d; threadId = %d; blockDim = %d!\n", blockIdx.x, threadIdx.x, blockDim.x);
}

int main() {
    // Input values
    float host_a = 3.0f;
    float host_b = 4.0f;

    // Result variable on the host
    float host_result;

    // Result variable on the device
    float *device_result;
    hipMalloc((void**)&device_result, sizeof(float));

    // Launch the add kernel with one block and one thread
    // <<<NumBlocks, BlockSize>>>   contem o numero de blocos e o numero de threads por bloco, respectivamente
    addKernel<<<2, 3>>>(host_a, host_b, device_result);

    // Copy the result back to the host
    hipMemcpy(&host_result, device_result, sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    printf("Result of %.2f + %.2f on GPU: %.2f\n", host_a, host_b, host_result);

    // Free memory
    hipFree(device_result);

    return 0;
}
